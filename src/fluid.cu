#include "hip/hip_runtime.h"
#include <omp.h>
#include <cmath>
#include <cstdint>
#include <cstdlib>
#include <iostream>

#include <thrust/device_ptr.h>
#include <thrust/reduce.h>

#include "SDL_rect.h"

#include "config_parser.hpp"
#include "fluid.cuh"
#include "helper.cuh"

__device__ void Fluid::trace(int i,
                             int j,
                             float d_t,
                             int* trace_line_x,
                             int* trace_line_y,
                             int trace_length) const {
  Vector2d<float> position = this->get_center_position(i, j);
  trace_line_x[0] = static_cast<int>(round(position.get_x()));
  trace_line_y[0] =
      this->height - 1 - static_cast<int>(round(position.get_y()));

  for (int k = 1; k < trace_length; k++) {
    auto x = position.get_x();
    auto y = position.get_y();
    Vector2d<float> velocity = this->get_general_velocity(x, y);
    position = position + velocity * d_t;
    trace_line_x[k] = static_cast<int>(round(position.get_x()));
    trace_line_y[k] =
        this->height - 1 - static_cast<int>(round(position.get_y()));
  }
}

__device__ __host__ bool Fluid::is_edge(int i, int j) const {
  return i == 0 || j == 0 || i == this->width - 1 || j == this->height - 1;
}
Fluid::Fluid(Config config)
    : width(config.sim.width),
      height(config.sim.height),
      o(config.sim.projection.o),
      n(config.sim.projection.n),
      cell_size(config.sim.cell_size),
      density(config.fluid.density),
      g(config.sim.physics.g),
      drag_coeff(config.fluid.drag_coeff),
      wind_tunnel_speed(config.sim.wind_tunnel.speed),
      wind_tunnel_height(config.sim.wind_tunnel.pipe_height),
      wind_tunnel_smoke_length(config.sim.wind_tunnel.smoke_length),
      wind_tunnel_smoke(config.sim.wind_tunnel.smoke),
      wind_tunnel_smoke_count(config.sim.wind_tunnel.smoke_count),
      wind_tunnel_smoke_height(config.sim.wind_tunnel.smoke_height),
      enable_smoke_decay(config.sim.smoke.enable_decay),
      enable_smoke(config.sim.enable_smoke),
      enable_pressure(config.sim.enable_pressure),
      smoke_decay_rate(config.sim.smoke.decay_rate),
      enable_interactive(config.sim.enable_interactive),
      viscosity(config.fluid.viscosity) {
  int grid_x =
      std::ceil(static_cast<float>(width) / config.thread.cuda.block_size_x);
  int grid_y =
      std::ceil(static_cast<float>(height) / config.thread.cuda.block_size_y);
  this->kernel_grid_dim = dim3(grid_x, grid_y, 1);
  this->kernel_block_dim =
      dim3(config.thread.cuda.block_size_x, config.thread.cuda.block_size_y, 1);
  this->alloc_device_memory();
  this->init_device_memory(config);
}

Fluid::~Fluid() {
  hipFree(this->d_this);
  hipFree(this->d_pressure);
  hipFree(this->d_vel_x);
  hipFree(this->d_vel_y);
  hipFree(this->d_smoke);
  hipFree(this->d_vel_x_buffer);
  hipFree(this->d_vel_y_buffer);
  hipFree(this->d_smoke_buffer);
  hipFree(this->d_is_solid);
  hipFree(this->d_total_s);
}

void Fluid::alloc_device_memory() {
  hipMalloc(&this->d_this, sizeof(Fluid));
  hipMalloc(&this->d_pressure, this->width * this->height * sizeof(float));
  hipMalloc(&this->d_vel_x, this->width * this->height * sizeof(float));
  hipMalloc(&this->d_vel_y, this->width * this->height * sizeof(float));
  hipMalloc(&this->d_smoke, this->width * this->height * sizeof(float));
  hipMalloc(&this->d_vel_x_buffer, this->width * this->height * sizeof(float));
  hipMalloc(&this->d_vel_y_buffer, this->width * this->height * sizeof(float));
  hipMalloc(&this->d_smoke_buffer, this->width * this->height * sizeof(float));
  hipMalloc(&this->d_is_solid, this->width * this->height * sizeof(int));
  hipMalloc(&this->d_total_s, this->width * this->height * sizeof(int));
}

void Fluid::init_device_memory(Config config) {
  int* is_solid =
      static_cast<int*>(std::malloc(this->height * this->width * sizeof(int)));
  int* total_s =
      static_cast<int*>(std::malloc(this->height * this->width * sizeof(int)));
  float* vel_x = static_cast<float*>(
      std::calloc(this->width * this->height, sizeof(float)));
  float* vel_y = static_cast<float*>(
      std::calloc(this->width * this->height, sizeof(float)));
  float* smoke = static_cast<float*>(
      std::calloc(this->width * this->height, sizeof(float)));

  for (auto i = 0; i < this->width; i++) {
    for (auto j = 0; j < this->height; j++) {
      is_solid[indx(i, j)] =
          (i == 0 or j == 0 or j == this->height - 1 or
           (!config.sim.enable_drain and i == this->width - 1) or
           (config.sim.obstacle.enable and
                std::sqrt(std::pow((i - config.sim.obstacle.center_x), 2) +
                          std::pow((j - config.sim.obstacle.center_y), 2)) <
                    config.sim.obstacle.radius or
            (i < config.sim.wind_tunnel.pipe_length &&
             (j == this->height / 2 - config.sim.wind_tunnel.pipe_height / 2 -
                       1 or
              j == this->height / 2 + config.sim.wind_tunnel.pipe_height / 2 +
                       1))));
    }
  }
  for (auto i = 0; i < this->width; i++) {
    for (auto j = 0; j < this->height; j++) {
      if (index_is_valid(i - 1, j) and is_solid[indx(i - 1, j)] == 0) {
        total_s[indx(i, j)]++;
      }
      if (index_is_valid(i + 1, j) and is_solid[indx(i + 1, j)] == 0) {
        total_s[indx(i, j)]++;
      }
      if (index_is_valid(i, j - 1) and is_solid[indx(i, j - 1)] == 0) {
        total_s[indx(i, j)]++;
      }
      if (index_is_valid(i, j + 1) and is_solid[indx(i, j + 1)] == 0) {
        total_s[indx(i, j)]++;
      }
    }
  }

  hipMemcpy(this->d_is_solid, is_solid,
             this->width * this->height * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(this->d_total_s, total_s, this->width * this->height * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(this->d_vel_x, vel_x, this->width * this->height * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(this->d_vel_y, vel_y, this->width * this->height * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(this->d_smoke, smoke, this->width * this->height * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(d_this, this, sizeof(Fluid), hipMemcpyHostToDevice);

  std::free(is_solid);
  std::free(total_s);
  std::free(vel_x);
  std::free(vel_y);
  std::free(smoke);
}

__device__ __host__ int Fluid::indx(int i, int j) const {
  return (this->height - j - 1) * this->width + i;
}

__global__ void apply_diffusion_kernel(Fluid* d_fluid, float d_t) {
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= d_fluid->width - 1 or j >= d_fluid->height - 1 or i <= 0 or j <= 0) {
    return;
  }
  d_fluid->apply_diffusion_at(i, j, d_t);
}

__device__ void Fluid::apply_diffusion_at(int i, int j, float d_t) {
  float a = this->viscosity * d_t / square(cell_size);
  d_vel_x[indx(i, j)] =
      (d_vel_x[indx(i, j)] +
       a * (d_vel_x[indx(i - 1, j)] + d_vel_x[indx(i + 1, j)] +
            d_vel_x[indx(i, j - 1)] + d_vel_x[indx(i, j + 1)])) /
      (1 + 4 * a);
}

void Fluid::apply_diffusion(float d_t) {
  for (int _; _ < this->n; _++) {
    apply_diffusion_kernel<<<this->kernel_grid_dim, this->kernel_block_dim>>>(
        d_this, d_t);
  }
}

__device__ float Fluid::get_divergence(int i, int j) const {
  auto u = this->d_vel_x[indx(i, j)];
  auto v = this->d_vel_y[indx(i, j)];
  auto top_v = this->d_vel_y[indx(i, j + 1)];
  auto right_u = this->d_vel_x[indx(i + 1, j)];

  auto divergence = right_u - u + top_v - v;

  return divergence;
}

__global__ void zero_pressure_kernel(Fluid* d_fluid) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i >= d_fluid->width or j >= d_fluid->height) {
    return;
  }
  d_fluid->zero_pressure_at(i, j);
}

__device__ void Fluid::zero_pressure_at(int i, int j) {
  this->d_pressure[indx(i, j)] = 0;
}

void Fluid::zero_pressure() {
  zero_pressure_kernel<<<this->kernel_grid_dim, this->kernel_block_dim>>>(
      d_this);
}

__device__ void Fluid::update_pressure_at(int i,
                                          int j,
                                          float velocity_diff,
                                          float d_t) {
  this->d_pressure[indx(i, j)] +=
      velocity_diff * this->density * this->cell_size / d_t;
}

__device__ void Fluid::apply_projection_at(int i, int j, float d_t) {
  if (this->d_is_solid[indx(i, j)]) {
    return;
  }

  auto u = this->d_vel_x[indx(i, j)];
  auto v = this->d_vel_y[indx(i, j)];
  auto top_v = this->d_vel_y[indx(i, j + 1)];
  auto right_u = this->d_vel_x[indx(i + 1, j)];

  auto divergence = right_u - u + top_v - v;
  auto s = this->d_total_s[indx(i, j)];
  auto velocity_diff = this->o * (divergence / s);

  if (this->enable_pressure) {
    this->update_pressure_at(i, j, velocity_diff, d_t);
  }

  if (not this->d_is_solid[indx(i - 1, j)]) {
    this->d_vel_x[indx(i, j)] += velocity_diff;
  }

  if (not this->d_is_solid[indx(i + 1, j)]) {
    this->d_vel_x[indx(i + 1, j)] -= velocity_diff;
  }

  if (not this->d_is_solid[indx(i, j - 1)]) {
    this->d_vel_y[indx(i, j)] += velocity_diff;
  }

  if (not this->d_is_solid[indx(i, j + 1)]) {
    this->d_vel_y[indx(i, j + 1)] -= velocity_diff;
  }
}

__global__ void apply_projection_even_kernel(Fluid* d_fluid, float d_t) {
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int i = (blockIdx.x * blockDim.x + threadIdx.x) * 2 + (j % 2);
  if (i >= d_fluid->width - 1 or j >= d_fluid->height - 1 or i <= 0 or j <= 0) {
    return;
  }
  d_fluid->apply_projection_at(i, j, d_t);
}

__global__ void apply_projection_odd_kernel(Fluid* d_fluid, float d_t) {
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int i = (blockIdx.x * blockDim.x + threadIdx.x) * 2 + ((j + 1) % 2);
  if (i >= d_fluid->width - 1 or j >= d_fluid->height - 1 or i <= 0 or j <= 0) {
    return;
  }
  d_fluid->apply_projection_at(i, j, d_t);
}

void Fluid::apply_projection(float d_t) {
  int grid_x =
      std::ceil(static_cast<float>(this->width) / this->kernel_block_dim.x / 2);
  int grid_y =
      std::ceil(static_cast<float>(this->height) / this->kernel_block_dim.y);
  auto grid_dim = dim3(grid_x, grid_y);

  for (int _ = 0; _ < this->n; _++) {
    apply_projection_even_kernel<<<grid_dim, this->kernel_block_dim>>>(d_this,
                                                                       d_t);
    apply_projection_odd_kernel<<<grid_dim, this->kernel_block_dim>>>(d_this,
                                                                      d_t);
  }
}

__global__ void apply_external_forces_kernel(Source source,
                                             Fluid* d_fluid,
                                             float d_t) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i >= d_fluid->width or j >= d_fluid->height) {
    return;
  }
  d_fluid->apply_external_forces_at(source, i, j, d_t);
}

__device__ void Fluid::apply_external_forces_at(Source source,
                                                int i,
                                                int j,
                                                float d_t) {
  int smoke_spacing =
      (this->wind_tunnel_height -
       this->wind_tunnel_smoke_count * this->wind_tunnel_smoke_height) /
      (this->wind_tunnel_smoke_count - 1);

  if (i <= this->wind_tunnel_smoke_length and i != 0 &&
      j >= this->height / 2 - this->wind_tunnel_height / 2 &&
      j <= this->height / 2 + this->wind_tunnel_height / 2) {
    this->d_vel_x[indx(i, j)] = this->wind_tunnel_speed;
    if ((this->wind_tunnel_smoke_count == 1 &&
         j >= this->height / 2 - this->wind_tunnel_smoke_height / 2 &&
         j <= this->height / 2 + this->wind_tunnel_smoke_height / 2) ||
        (this->wind_tunnel_smoke_count != 1 &&
         (this->height / 2 + this->wind_tunnel_height / 2 - j) %
                 (smoke_spacing + this->wind_tunnel_smoke_height) <
             this->wind_tunnel_smoke_height)) {
      this->d_smoke[indx(i, j)] = this->wind_tunnel_smoke;
    }
  }
  if (this->drag_coeff != 0) {
    float damping = expf(-this->drag_coeff * d_t);
    this->d_vel_x[indx(i, j)] *= damping;
    this->d_vel_y[indx(i, j)] *= damping;
  }
  if (source.active && square(i - source.position.get_x()) +
                               square(j - source.position.get_y()) <
                           square(40)) {
    if (source.smoke) {
      this->d_smoke[indx(i, j)] = source.smoke;
    }
    float x_speed_modifier = i - source.position.get_x();
    float y_speed_modifier = j - source.position.get_y();
    this->d_vel_x[indx(i, j)] += source.velocity * x_speed_modifier;
    this->d_vel_y[indx(i, j)] += source.velocity * y_speed_modifier;
  }

  if (!this->d_is_solid[indx(i, j - 1)]) {
    this->d_vel_y[indx(i, j)] += this->g * d_t;
  }
}

void Fluid::apply_external_forces(Source source, float d_t) {
  apply_external_forces_kernel<<<this->kernel_grid_dim,
                                 this->kernel_block_dim>>>(source, d_this, d_t);
}

__device__ __host__ bool Fluid::index_is_valid(int i, int j) const {
  return i < this->width and j < this->height and i >= 0 and j >= 0;
}

__device__ bool Fluid::is_valid_fluid(int i, int j) const {
  return index_is_valid(i, j) and not this->d_is_solid[indx(i, j)];
}

__device__ Vector2d<float> Fluid::get_vertical_edge_velocity(int i,
                                                             int j) const {
  auto u = this->d_vel_x[indx(i, j)];

  auto avg_v = this->d_vel_y[indx(i, j)];
  int count = 1;

  if (is_valid_fluid(i - 1, j + 1)) {
    avg_v += this->d_vel_y[indx(i - 1, j + 1)];
    count++;
  }

  if (is_valid_fluid(i, j + 1)) {
    avg_v += this->d_vel_y[indx(i, j + 1)];
    count++;
  }

  if (is_valid_fluid(i - 1, j)) {
    avg_v += this->d_vel_y[indx(i - 1, j)];
    count++;
  }

  avg_v /= count;

  return Vector2d<float>(u, avg_v);
}

__device__ Vector2d<float> Fluid::get_horizontal_edge_velocity(int i,
                                                               int j) const {
  auto v = this->d_vel_y[indx(i, j)];

  float avg_u = this->d_vel_x[indx(i, j)];
  int count = 1;

  if (is_valid_fluid(i + 1, j)) {
    avg_u += this->d_vel_x[indx(i + 1, j)];
    count++;
  }

  if (is_valid_fluid(i, j - 1)) {
    avg_u += this->d_vel_x[indx(i, j - 1)];
    count++;
  }

  if (is_valid_fluid(i + 1, j - 1)) {
    avg_u += this->d_vel_x[indx(i + 1, j - 1)];
    count++;
  }

  avg_u /= count;

  return Vector2d<float>(avg_u, v);
}

__device__ float Fluid::get_general_velocity_y(float x, float y) const {
  int i = x / this->cell_size;
  int j = y / this->cell_size;

  if (not this->is_valid_fluid(i, j)) {
    return 0;
  }

  float in_x = x - i * this->cell_size;
  float in_y = y - j * this->cell_size;

  float avg_v = 0;

  // take average with the left cell
  if (in_x < this->cell_size / 2.0) {
    float d_x = this->cell_size / 2.0 - in_x;
    float w_x = 1 - d_x / this->cell_size;
    float w_y = 1 - in_y / this->cell_size;

    if (this->is_valid_fluid(i, j)) {
      avg_v += w_y * w_x * this->d_vel_y[indx(i, j)];
    }

    if (this->is_valid_fluid(i - 1, j)) {
      avg_v += w_y * (1 - w_x) * this->d_vel_y[indx(i - 1, j)];
    }

    if (this->is_valid_fluid(i - 1, j + 1)) {
      avg_v += (1 - w_y) * (1 - w_x) * this->d_vel_y[indx(i - 1, j + 1)];
    }

    if (this->is_valid_fluid(i, j + 1)) {
      avg_v += (1 - w_y) * w_x * this->d_vel_y[indx(i, j + 1)];
    }
  }
  // take average with the right cell
  else {
    float d_x = in_x - this->cell_size / 2.0;
    float w_x = 1 - d_x / this->cell_size;
    float w_y = 1 - in_y / this->cell_size;

    if (this->is_valid_fluid(i, j)) {
      avg_v += w_y * w_x * this->d_vel_y[indx(i, j)];
    }

    if (this->is_valid_fluid(i, j + 1)) {
      avg_v += (1 - w_y) * w_x * this->d_vel_y[indx(i, j + 1)];
    }

    if (this->is_valid_fluid(i + 1, j + 1)) {
      avg_v += (1 - w_y) * (1 - w_x) * this->d_vel_y[indx(i + 1, j + 1)];
    }

    if (this->is_valid_fluid(i + 1, j)) {
      avg_v += w_y * (1 - w_x) * this->d_vel_y[indx(i + 1, j)];
    }
  }

  return avg_v;
}

__device__ float Fluid::get_general_velocity_x(float x, float y) const {
  int i = x / this->cell_size;
  int j = y / this->cell_size;

  if (not this->is_valid_fluid(i, j)) {
    return 0;
  }

  float in_x = x - i * this->cell_size;
  float in_y = y - j * this->cell_size;

  float avg_u = 0;

  // take average with the bottom cell
  if (in_y <= this->cell_size / 2.0) {
    float d_y = this->cell_size / 2.0 - in_y;
    float w_x = 1 - in_x / this->cell_size;
    float w_y = 1 - d_y / this->cell_size;

    if (this->is_valid_fluid(i, j)) {
      avg_u += w_y * w_x * this->d_vel_x[indx(i, j)];
    }

    if (this->is_valid_fluid(i + 1, j)) {
      avg_u += w_y * (1 - w_x) * this->d_vel_x[indx(i + 1, j)];
    }

    if (this->is_valid_fluid(i, j - 1)) {
      avg_u += (1 - w_y) * w_x * this->d_vel_x[indx(i, j - 1)];
    }

    if (this->is_valid_fluid(i + 1, j - 1)) {
      avg_u += (1 - w_y) * (1 - w_x) * this->d_vel_x[indx(i + 1, j - 1)];
    }
  }

  // take average with the top cell
  else {
    float d_y = in_y - this->cell_size / 2.0;
    float w_x = 1 - in_x / this->cell_size;
    float w_y = 1 - d_y / this->cell_size;

    if (this->is_valid_fluid(i, j)) {
      avg_u += w_y * w_x * this->d_vel_x[indx(i, j)];
    }

    if (this->is_valid_fluid(i, j + 1)) {
      avg_u += (1 - w_y) * w_x * this->d_vel_x[indx(i, j + 1)];
    }

    if (this->is_valid_fluid(i + 1, j)) {
      avg_u += w_y * (1 - w_x) * this->d_vel_x[indx(i + 1, j)];
    }

    if (this->is_valid_fluid(i + 1, j + 1)) {
      avg_u += (1 - w_y) * (1 - w_x) * this->d_vel_x[indx(i + 1, j + 1)];
    }
  }

  return avg_u;
}

__device__ Vector2d<float> Fluid::get_general_velocity(float x, float y) const {
  float u = this->get_general_velocity_x(x, y);
  float v = this->get_general_velocity_y(x, y);
  return Vector2d<float>(u, v);
}

__device__ Vector2d<float> Fluid::get_center_position(int i, int j) const {
  return Vector2d<float>((i + 0.5) * this->cell_size,
                         (j + 0.5) * this->cell_size);
}

__device__ Vector2d<float> Fluid::get_u_position(int i, int j) const {
  return Vector2d<float>(i * this->cell_size, (j + 0.5) * this->cell_size);
}

__device__ Vector2d<float> Fluid::get_v_position(int i, int j) const {
  return Vector2d<float>((i + 0.5) * this->cell_size, j * this->cell_size);
}

__device__ void Fluid::apply_smoke_advection_at(int i, int j, float d_t) {
  Vector2d<float> current_pos = this->get_center_position(i, j);
  Vector2d<float> current_velocity =
      this->get_general_velocity(current_pos.get_x(), current_pos.get_y());
  auto prev_pos = current_pos - current_velocity * d_t;
  float new_smoke = interpolate_smoke(prev_pos.get_x(), prev_pos.get_y());
  this->d_smoke_buffer[indx(i, j)] = new_smoke;
}

__device__ void Fluid::update_smoke_advection_at(int i, int j, float d_t) {
  this->d_smoke[indx(i, j)] = this->d_smoke_buffer[indx(i, j)];
}

__global__ void apply_smoke_advection_kernel(Fluid* d_fluid, float d_t) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i >= d_fluid->width or j >= d_fluid->height) {
    return;
  }
  d_fluid->apply_smoke_advection_at(i, j, d_t);
}

__global__ void update_smoke_advection_kernel(Fluid* d_fluid, float d_t) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i >= d_fluid->width or j >= d_fluid->height) {
    return;
  }
  d_fluid->update_smoke_advection_at(i, j, d_t);
}

void Fluid::apply_smoke_advection(float d_t) {
  apply_smoke_advection_kernel<<<this->kernel_grid_dim,
                                 this->kernel_block_dim>>>(d_this, d_t);
  update_smoke_advection_kernel<<<this->kernel_grid_dim,
                                  this->kernel_block_dim>>>(d_this, d_t);
}

__device__ void Fluid::apply_velocity_advection_at(int i, int j, float d_t) {
  Vector2d<float> current_pos = this->get_u_position(i, j);
  Vector2d<float> current_velocity = this->get_vertical_edge_velocity(i, j);
  auto prev_pos = current_pos - current_velocity * d_t;
  float new_velocity =
      this->get_general_velocity_x(prev_pos.get_x(), prev_pos.get_y());
  this->d_vel_x_buffer[indx(i, j)] = new_velocity;

  current_pos = this->get_v_position(i, j);
  current_velocity = this->get_horizontal_edge_velocity(i, j);
  prev_pos = current_pos - current_velocity * d_t;
  new_velocity =
      this->get_general_velocity_y(prev_pos.get_x(), prev_pos.get_y());
  this->d_vel_y_buffer[indx(i, j)] = new_velocity;
}

__device__ void Fluid::update_velocity_advection_at(int i, int j, float d_t) {
  this->d_vel_x[indx(i, j)] = this->d_vel_x_buffer[indx(i, j)];
  this->d_vel_y[indx(i, j)] = this->d_vel_y_buffer[indx(i, j)];
}

__global__ void apply_velocity_advection_kernel(Fluid* d_fluid, float d_t) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i >= d_fluid->width or j >= d_fluid->height) {
    return;
  }
  d_fluid->apply_velocity_advection_at(i, j, d_t);
}

__global__ void update_velocity_advection_kernel(Fluid* d_fluid, float d_t) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i >= d_fluid->width or j >= d_fluid->height) {
    return;
  }
  d_fluid->update_velocity_advection_at(i, j, d_t);
}

void Fluid::apply_velocity_advection(float d_t) {
  apply_velocity_advection_kernel<<<this->kernel_grid_dim,
                                    this->kernel_block_dim>>>(d_this, d_t);
  update_velocity_advection_kernel<<<this->kernel_grid_dim,
                                     this->kernel_block_dim>>>(d_this, d_t);
}

__device__ float Fluid::interpolate_smoke(float x, float y) const {
  float normalized_x = x / this->cell_size;
  float normalized_y = y / this->cell_size;
  float shifted_x = normalized_x - 0.5;
  float shifted_y = normalized_y - 0.5;

  int i = shifted_x;
  int j = shifted_y;

  float smoke_00 = this->d_smoke[indx(i, j)];
  float smoke_10 = this->d_smoke[indx(i + 1, j)];
  float smoke_01 = this->d_smoke[indx(i, j + 1)];
  float smoke_11 = this->d_smoke[indx(i + 1, j + 1)];

  float wx_0 = (i + 1) - shifted_x;
  float wy_0 = (j + 1) - shifted_y;
  float wx_1 = shifted_x - i;
  float wy_1 = shifted_y - j;

  float w_00 = wx_0 * wy_0;
  float w_01 = wx_0 * wy_1;
  float w_10 = wx_1 * wy_0;
  float w_11 = wx_1 * wy_1;

  return w_00 * smoke_00 + w_10 * smoke_10 + w_01 * smoke_01 + w_11 * smoke_11;
}

// ? review this
// ? review logic
__device__ void Fluid::apply_extrapolation_at(int i, int j) {
  if (j == 0) {
    this->d_vel_x[indx(i, j)] = this->d_vel_x[indx(i, j + 1)];
  } else if (j == this->height - 1) {
    this->d_vel_x[indx(i, j)] = this->d_vel_x[indx(i, j - 1)];
  }
  if (i == 0) {
    this->d_vel_y[indx(i, j)] = this->d_vel_y[indx(i + 1, j)];
  } else if (i == this->width - 1) {
    this->d_vel_y[indx(i, j)] = this->d_vel_y[indx(i - 1, j)];
  }
}

__global__ void apply_extrapolation_kernel(Fluid* d_fluid) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i >= d_fluid->width or j >= d_fluid->height) {
    return;
  }
  d_fluid->apply_extrapolation_at(i, j);
}

void Fluid::apply_extrapolation() {
  apply_extrapolation_kernel<<<this->kernel_grid_dim, this->kernel_block_dim>>>(
      d_this);
}

__global__ void decay_smoke_kernel(Fluid* d_fluid, float d_t) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i >= d_fluid->width or j >= d_fluid->height) {
    return;
  }
  d_fluid->decay_smoke_at(i, j, d_t);
}

__device__ void Fluid::decay_smoke_at(int i, int j, float d_t) {
  if (this->enable_smoke_decay)
    this->d_smoke[this->indx(i, j)] = max(
        this->d_smoke[this->indx(i, j)] - this->smoke_decay_rate * d_t, 0.0);
}

void Fluid::decay_smoke(float d_t) {
  decay_smoke_kernel<<<this->kernel_grid_dim, this->kernel_block_dim>>>(d_this,
                                                                        d_t);
}

// ? put the whole thing into a graph
void Fluid::update(Source source, float d_t) {
  this->apply_external_forces(source, d_t);
  if (this->enable_pressure)
    this->zero_pressure();
  if (this->viscosity != 0) {
    this->apply_diffusion(d_t);
  }
  this->apply_projection(d_t);
  if (this->enable_pressure) {
    thrust::device_ptr<float> device_pressure =
        thrust::device_pointer_cast(this->d_pressure);
    this->min_pressure = thrust::reduce(
        device_pressure, device_pressure + (this->width * this->height),
        std::numeric_limits<float>::infinity(), thrust::minimum<float>());
    this->max_pressure = thrust::reduce(
        device_pressure, device_pressure + (this->width * this->height),
        -std::numeric_limits<float>::infinity(), thrust::maximum<float>());
  }
  this->apply_extrapolation();
  this->apply_velocity_advection(d_t);
  if (this->enable_smoke && this->wind_tunnel_smoke != 0) {
    this->apply_smoke_advection(d_t);
    this->decay_smoke(d_t);
  }
  hipDeviceSynchronize();
}
